#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/communicator/multi_process_data_parallel_communicator.hpp>

#include <algorithm>
#include <cstdlib>
#include <memory>

#include "mpi.h"

namespace nbla {

using std::make_shared;

template <typename T>
__global__ void kernel_divide_inplace(const int size, const int n_devices,
                                      T *dw) {
  NBLA_CUDA_KERNEL_LOOP(i, size) { dw[i] /= n_devices; }
}

__global__ void kernel_null() {}

template <typename T>
MultiProcessDataParallelCommunicatorNccl<
    T>::MultiProcessDataParallelCommunicatorNccl(const Context &ctx)
    : MultiProcessDataParallelCommunicator<T>(ctx) {
  mpi_initialized_ = false;
}

template <typename T>
MultiProcessDataParallelCommunicatorNccl<
    T>::~MultiProcessDataParallelCommunicatorNccl() {
  if (this->initialized_) {
    for (int i = 0; i < streams_.size(); ++i) {
      NBLA_CUDA_CHECK(hipStreamDestroy(streams_[i]));
    }
    ncclCommDestroy(comm_);
  }
  if (mpi_initialized_) {
    MPI_Finalize();
  }
}

template <typename T>
bool MultiProcessDataParallelCommunicatorNccl<T>::mpi_initialized_;

template <typename T> void MultiProcessDataParallelCommunicatorNccl<T>::init() {
  Communicator::init();
  try {
    // MPI init
    if (!mpi_initialized_) {
      int argc = 0;
      char **argv = NULL;
      int requiredThreadLevelSupport = MPI_THREAD_SERIALIZED;
      int provided;
      MPI_Init_thread(&argc, &argv, requiredThreadLevelSupport, &provided);
      if (provided != requiredThreadLevelSupport)
        NBLA_ERROR(error_code::target_specific, "MPI_Init_thread failed.");
      mpi_initialized_ = true;
    }
    // Create comm, set size, and rank
    MPI_Comm mpi_comm;
    MPI_Comm_dup(MPI_COMM_WORLD, &mpi_comm);
    MPI_Comm_size(mpi_comm, &this->size_);
    MPI_Comm_rank(mpi_comm, &this->rank_);
    device_id_ = this->rank_;

    // We have to set our device before NCCL init
    cuda_set_device(device_id_);
    MPI_Barrier(mpi_comm);

    // Exchange comm_id_ among processes
    ncclGetUniqueId(&comm_id_);
    MPI_Bcast(&comm_id_, NCCL_UNIQUE_ID_BYTES, MPI_CHAR, 0, mpi_comm);
    MPI_Comm_free(&mpi_comm);

    // Nccl Init
    ncclResult_t ret =
        ncclCommInitRank(&comm_, this->size_, comm_id_, this->rank_);
    if (ret != ncclSuccess) {
      NBLA_ERROR(error_code::target_specific, "ncclCommInitRank failed.");
    }

    // Create streams
    for (int i = 0; i < streams_.size(); ++i) {
      // Stream
      hipStream_t stream;
      NBLA_CUDA_CHECK(hipStreamCreate(&stream));
      streams_[i] = stream;
    }

    this->initialized_ = true;
  } catch (...) {
    NBLA_ERROR(error_code::unclassified, "Communicator init failed.");
  }
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::reduce(bool division) {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU ireduce is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::allreduce(bool division,
                                                            bool inplace) {
  // TODO: currently nnabla uses default stream for computation.
  // The following logic relies on that, so if nnabla uses another stream for
  // computation,
  // we have to issue null kernel to the default stream at the beginning of this
  // method
  // and at the end of this method for using the implicit synchronization
  // technique for
  // main thread not to wait for a result of a kernel call.

  // Once sync to prevent the hang where the memcpy occurs during the allreduce.
  this->sync_all_params();

  if (inplace) { // in-place
    Context ctx = this->contexts_[0];

    auto func_named_param = this->device_func_named_param_[0];
    auto size = func_named_param.size();

    int k = 0;
    for (auto elm : func_named_param) { // function-loop
      VariablePtr vp = elm.second;
      auto n_param = vp->size();

      const T *dw0 = vp->get_grad_pointer<T>(ctx);
      T *dw1 = vp->cast_grad_and_get_pointer<T>(ctx);
      int stream_id = k % num_streams_;
      ncclResult_t ret = ncclAllReduce(dw0, dw1, n_param, ncclFloat,
                                       ncclSum, // TODO: address ncclFloat
                                       comm_, streams_[stream_id]);
      if (ret != ncclSuccess) {
        NBLA_ERROR(error_code::target_specific, "ncclAllReduce fails with %d.",
                   ret);
      }
      k++;
    }
    // Divide using the same streams
    divide_by_num_divices(division);

  } else { // out-of-place. use a large array.
    Context ctx = this->contexts_[0];
    shared_ptr<CudaCachedArray> arr_buff = // TODO: address 16 bits also here?
        make_shared<CudaCachedArray>(this->total_params_, get_dtype<T>(), ctx);

    T *buff = arr_buff->pointer<T>();
    T *buff_start = buff;
    auto func_named_param = this->device_func_named_param_[0];
    Size_t type_size = sizeof(T);
    int k = 0;

    // 1. copy inside device
    for (auto elm : func_named_param) {
      VariablePtr vp = elm.second;
      const T *dw = vp->get_grad_pointer<T>(ctx);
      auto n_param = vp->size();
      int stream_id = k % num_streams_;
      hipMemcpyAsync(buff, dw, type_size * n_param, hipMemcpyDeviceToDevice,
                      streams_[stream_id]);
      buff += n_param;
      k++;
    }

    // 2. allreduce
    ncclResult_t ret =
        ncclAllReduce(buff_start, buff_start, this->total_params_,
                      ncclFloat,          // TODO: address ncclFloat
                      ncclSum, comm_, 0); // use default stream

    if (ret != ncclSuccess) {
      NBLA_ERROR(error_code::target_specific, "ncclAllReduce fails with %d.",
                 ret);
    }

    // 3. divide
    if (division) {
      // use default stream
      NBLA_CUDA_LAUNCH_KERNEL_IN_STREAM(kernel_divide_inplace, 0,
                                        this->total_params_, this->size_,
                                        buff_start);
    }

    // 4. copy back inside device
    buff = buff_start;
    k = 0;
    for (auto elm : func_named_param) {
      VariablePtr vp = elm.second;
      T *dw = vp->cast_grad_and_get_pointer<T>(ctx);
      auto n_param = vp->size();
      int stream_id = k % num_streams_;
      hipMemcpyAsync(dw, buff, type_size * n_param, hipMemcpyDeviceToDevice,
                      streams_[stream_id]);
      buff += n_param;
      k++;
    }
  }
  // no need to call null kernel since nnabla uses default stream currently.
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::reducescatter(bool division) {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU reducescatter is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::bcast() {
  NBLA_ERROR(error_code::not_implemented, "CUDA GPU bcast is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::allgather() {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU allgather is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::reduce_async(bool division) {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU reduce_async is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::allreduce_async(
    bool division, bool inplace) {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU allreduce_async is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::reducescatter_async(
    bool division) {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU reducescatter_async is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::bcast_async() {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU bcast_async is not implemented.")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::allgather_async() {
  NBLA_ERROR(error_code::not_implemented,
             "CUDA GPU allgather_async is not implemented.")
}

template <typename T>
vector<string>
MultiProcessDataParallelCommunicatorNccl<T>::allowed_array_classes() {
  NBLA_ERROR(error_code::not_implemented,
             "Derived class of MultiProcessDataParallelCommunicatorNccl must "
             "implement allowed_array_classes().")
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<
    T>::wait_by_device_synchronization() {
  cuda_device_synchronize(device_id_);
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<
    T>::wait_by_streams_synchronization() {
  for (int i = 0; i < streams_.size(); ++i) {
    NBLA_CUDA_CHECK(hipStreamSynchronize(streams_[i]));
  }
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::divide_by_num_divices(
    bool division) {
  if (division) {
    Context ctx = this->contexts_[0];
    auto func_named_param = this->device_func_named_param_[0];
    int k = 0;
    for (auto elm : func_named_param) {
      VariablePtr vp = elm.second;
      T *dw = vp->cast_grad_and_get_pointer<T>(ctx);
      auto n_param = vp->size();
      int stream_id = k % num_streams_;
      NBLA_CUDA_LAUNCH_KERNEL_IN_STREAM(
          kernel_divide_inplace, streams_[stream_id], n_param, this->size_, dw);
      k++;
    }
  }
}

template <typename T>
void MultiProcessDataParallelCommunicatorNccl<T>::sync_all_params() {
  Context ctx = this->contexts_[0];
  auto func_named_param = this->device_func_named_param_[0];
  auto size = func_named_param.size();

  for (auto elm : func_named_param) { // function-loop
    VariablePtr vp = elm.second;

    // If the arrays are different, output the warning.
    this->check_array_class(ctx, vp);

    // Sync
    vp->get_grad_pointer<T>(ctx);
  }
}

template class MultiProcessDataParallelCommunicatorNccl<float>;
}
