#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// relu.cpp

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/relu.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_relu_forward(const int num, T *y, const T *x) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = max(T(0), x[idx]); }
}

template <typename T, bool accum = true>
__global__ void kernel_relu_backward(const int num, T *dx, const T *x,
                                     const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    dx[idx] = (accum ? dx[idx] : 0) + (x[idx] > 0 ? dy[idx] : 0);
  }
}

template <class T>
void ReLUCuda<T>::forward_impl(const Variables &inputs,
                               const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_relu_forward, size, y, x);
}

template <class T>
void ReLUCuda<T>::backward_impl(const Variables &inputs,
                                const Variables &outputs,
                                const vector<bool> &propagate_down,
                                const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  if (dx != dy && accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_relu_backward<T, true>), size, dx, x,
                                   dy);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_relu_backward<T, false>), size, dx,
                                   x, dy);
  }
}

// Template instantiation
template class ReLUCuda<float>;
}
