#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/function/prod.hpp>
#include <nbla/cuda/utils/device_reduce.cuh>
#include <nbla/cuda/utils/reduce_ops/prod.cuh>

namespace nbla {

template <typename T>
void ProdCuda<T>::forward_impl_reduce(const T *x, T *y, int outer_size,
                                      int reduction_size) {
  cuda_set_device(this->device_);
  // TODO: Auto tune.
  if (reduction_size / outer_size < 32) {
    reduce_2d_mixed_parallel(outer_size, reduction_size, ProdOp<T>(x, y));
    return;
  }

  // Get block reduce buffer
  auto fbuff = cuda_get_reduction_buffer<T>(reduction_size, this->ctx_);
  ProdOp<T> pre_op(x, fbuff.second);
  ProdOp<T> post_op(fbuff.second, y);
  reduce_2d_parallel_reduction(outer_size, reduction_size, pre_op, post_op);
}

template <typename T, bool accum>
__global__ void kernel_reduce_prod_backward(const int num, int reduction_size,
                                            const T *dy, const T *x, const T *y,
                                            T *dx) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    int o = idx / reduction_size;
    if (accum) {
      dx[idx] += x[idx] == 0 ? 0 : dy[o] * y[o] / x[idx];
    } else {
      dx[idx] = x[idx] == 0 ? 0 : dy[o] * y[o] / x[idx];
    }
  }
}

template <typename T>
void ProdCuda<T>::backward_impl_reduce_prod(const T *dy, const T *x, const T *y,
                                            T *dx, int outer_size,
                                            int reduction_size, bool accum) {
  cuda_set_device(this->device_);
  if (accum) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_prod_backward<T, true>),
                                   outer_size * reduction_size, reduction_size,
                                   dy, x, y, dx);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_prod_backward<T, false>),
                                   outer_size * reduction_size, reduction_size,
                                   dy, x, y, dx);
  }
}

// template instantiation
template class ProdCuda<float>;
}
