#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// reduce_sum.cu

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/reduce_sum.hpp>
#include <nbla/variable.hpp>

#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

namespace nbla {

template <typename T>
__global__ void kernel_reduce_sum_backward(const int num, T *dx, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { dx[idx] += *dy; }
}

template <class T>
void ReduceSumCuda<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  const Size_t size = inputs[0]->size();
  thrust::device_ptr<const T> x(inputs[0]->get_data_pointer<T>(this->ctx_));
  T sum = thrust::reduce(x, x + size, (T)0, thrust::plus<T>());
  hipMemcpy(y, &sum, sizeof(T), hipMemcpyHostToDevice);
}

template <class T>
void ReduceSumCuda<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const Size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_reduce_sum_backward, size, dx, dy);
}

template class ReduceSumCuda<float>;
}
