#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/solver/rmsprop.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_rmsprop_update(const int num, T *data, const T *grad,
                                      T *e_sqr_grad, const float lr,
                                      const float decay, const float eps) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    e_sqr_grad[idx] =
        e_sqr_grad[idx] * decay + grad[idx] * grad[idx] * (1 - decay);
    data[idx] -= lr * grad[idx] / (sqrt(e_sqr_grad[idx]) + eps);
  }
}

template <typename T>
__global__ void kernel_weight_decay(const int num, T *grad, const T *data,
                                    const float decay_rate) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { grad[idx] += decay_rate * data[idx]; }
}

template <typename T>
void RMSpropCuda<T>::update_impl(const string &key, VariablePtr param) {
  Size_t size = param->size();
  VariablePtr state = this->state_.at(key);
  T *e_sqr_grad = state->cast_data_and_get_pointer<T>(this->ctx_);
  const T *grad = param->get_grad_pointer<T>(this->ctx_);
  T *data = param->cast_data_and_get_pointer<T>(this->ctx_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_rmsprop_update, size, data, grad,
                                 e_sqr_grad, this->lr_, this->decay_,
                                 this->eps_);
}

NBLA_DEF_WEIGHT_DECAY(RMSpropCuda, weight_decay_cuda);

// Template instantiation
template class RMSpropCuda<float>;
}
