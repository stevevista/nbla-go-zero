#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/solver/momentum.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_momentum_update(const int num, T *data, const T *grad,
                                       T *v, const float lr,
                                       const float momentum) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    v[idx] = momentum * v[idx] + lr * grad[idx];
    data[idx] -= v[idx];
  }
}

template <typename T>
__global__ void kernel_weight_decay(const int num, T *grad, const T *data,
                                    const float decay_rate) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { grad[idx] += decay_rate * data[idx]; }
}

template <typename T>
void MomentumCuda<T>::update_impl(const string &key, VariablePtr param) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Size_t size = param->size();
  VariablePtr r_ = this->state_.at(key);
  const T *grad = param->get_grad_pointer<T>(this->ctx_);
  T *v = r_->cast_data_and_get_pointer<T>(this->ctx_);
  T *data = param->cast_data_and_get_pointer<T>(this->ctx_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_momentum_update, size, data, grad, v,
                                 this->lr_, this->momentum_);
}

NBLA_DEF_WEIGHT_DECAY(MomentumCuda, weight_decay_cuda);

// Template instantiation
template class MomentumCuda<float>;
}
