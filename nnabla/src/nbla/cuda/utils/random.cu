#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/utils/random.hpp>

namespace nbla {
// Generate rand(low, high) values from output of hiprandGenerateUniform.
// hiprandGenerateUniform returns random values in (0, 1], but we need [low,
// high).
template <typename T>
__global__ void kernel_rand_post_process(int size, T *dev_ptr, T low, T high) {
  NBLA_CUDA_KERNEL_LOOP(idx, size) {
    dev_ptr[idx] = (T(1) - dev_ptr[idx]) * (high - low) + low;
  }
}

static __global__ void kernel_randint_post_process(int size, int *dev_ptr,
                                                   int low, int high) {
  float *f_ptr = reinterpret_cast<float *>(dev_ptr);
  NBLA_CUDA_KERNEL_LOOP(idx, size) {
    dev_ptr[idx] = (1.0f - f_ptr[idx]) * (high - low) + low;
  }
}

template <>
void curand_generate_rand<float>(hiprandGenerator_t gen, float low, float high,
                                 float *dev_ptr, size_t size) {
  NBLA_CURAND_CHECK(hiprandGenerateUniform(gen, dev_ptr, size));
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_rand_post_process<float>), size,
                                 dev_ptr, low, high);
}

template <>
void curand_generate_rand<int>(hiprandGenerator_t gen, int low, int high,
                               int *dev_ptr, size_t size) {
  NBLA_CURAND_CHECK(
      hiprandGenerateUniform(gen, reinterpret_cast<float *>(dev_ptr), size));
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_randint_post_process, size, dev_ptr,
                                 low, high);
}
}
