#include "hip/hip_runtime.h"
// Copyright (C) 2015  Davis E. King (davis@dlib.net)
// License: Boost Software License   See LICENSE.txt for the full license.

#include "cuda_utils.h"
#include "cuda_dlib.h"


namespace dlib 
{ 
    namespace cuda 
    {

    // -----------------------------------------------------------------------------------

        void set_device (
            int dev
        )
        {
            CHECK_CUDA(hipSetDevice(dev));
        }

        int get_device (
        )
        {
            int dev = 0;
            CHECK_CUDA(hipGetDevice(&dev));
            return dev;
        }

        std::string get_device_name (
            int device
        )
        {
            hipDeviceProp_t props;
            CHECK_CUDA(hipGetDeviceProperties(&props, device));
            return props.name;
        }

        void set_current_device_blocking_sync(
        )
        {
            CHECK_CUDA(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        }

        int get_num_devices (
        )
        {
            int num_devices;
            CHECK_CUDA(hipGetDeviceCount(&num_devices));
            return num_devices;
        }

        bool can_access_peer (int device_id, int peer_device_id)
        {
            int can_access;
            CHECK_CUDA(hipDeviceCanAccessPeer(&can_access, device_id, peer_device_id));
            return can_access != 0;
        }
        bool can_access_peer (const tensor& device, const tensor& peer_device)
        {
            return can_access_peer(device.device_id(), peer_device.device_id());
        }

        void device_synchronize (int dev) 
        { 
            raii_set_device set_dev(dev);
            CHECK_CUDA(hipDeviceSynchronize());
        }
        void device_synchronize (const tensor& dev) { device_synchronize(dev.device_id()); }

        enable_peer_access::
        enable_peer_access(
            int device_id,
            int peer_device_id
        ) : call_disable(false), device_id(device_id), peer_device_id(peer_device_id)
        {
            raii_set_device set_dev(device_id);

            auto err = hipDeviceEnablePeerAccess(peer_device_id, 0);
            if (err == hipSuccess)
            {
                call_disable = true;
            }
            else if (err == hipErrorPeerAccessAlreadyEnabled)
            {
                // call hipGetLastError() to dispose of this error since we don't
                // care.
                auto err2 = hipGetLastError();
                if (err2 != hipErrorPeerAccessAlreadyEnabled)
                    CHECK_CUDA(err2);
            }
            else
            {
                CHECK_CUDA(err);
            }
        }


        enable_peer_access::
        ~enable_peer_access() noexcept(false)
        {
            if (call_disable)
            {
                raii_set_device set_dev(device_id);
                CHECK_CUDA(hipDeviceDisablePeerAccess(peer_device_id));
            }
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_inverse_norms(float* invnorms, const float* data, size_t nr, size_t nc, const float eps)
        {
            // initialize invnorms before we begin.
            for (auto i : grid_stride_range_y(0, nr))
                for (auto j : grid_stride_range(0, 1))
                    invnorms[i] = eps;
            __syncthreads();

            for (auto i : grid_stride_range_y(0, nr))
            {
                auto p = data + i*nc;
                float temp = 0;
                for (auto j : grid_stride_range(0, nc))
                    temp += p[j]*p[j];

                // and store the sum into invnorms[i]
                warp_reduce_atomic_add(invnorms[i], temp);
            }
            __syncthreads();

            for (auto i : grid_stride_range_y(0, nr))
                for (auto j : grid_stride_range(0, 1))
                    invnorms[i] = 1.0/std::sqrt(invnorms[i]);
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_add1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]+s2[i];
            }
        }

        __global__ void _cuda_add2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1+v2;
            }
        }

        void add (
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                launch_kernel(_cuda_add1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                // Otherwise, do the more complex version with bounds checking.
                launch_kernel(_cuda_add2,max_jobs(dest.size()),
                            dest.device(), src1.device(), src2.device(), 
                            dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                            src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                            src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                            );
            }

        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_scaled(float* d, const float* s, size_t n, float scale)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += scale*s[i]; 
            }
        }

        void add_scaled(
            tensor& dest,
            const float scale,
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.size()==src.size());
            launch_kernel(_cuda_add_scaled,max_jobs(dest.size()),dest.device(), src.device(), dest.size(), scale);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_cv_to_all_columns(float beta, float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = beta*dest[i] + alpha*src[i/stride];
            }
        }

        __global__ void _cuda_add_cv_to_all_columns_no_beta(float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = alpha*src[i/stride];
            }
        }

        void add_cv_to_all_columns(
            float beta, 
            tensor& dest, 
            float alpha, 
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.num_samples() == src.num_samples() && src.num_samples() == src.size());
            if (beta == 0)
                launch_kernel(_cuda_add_cv_to_all_columns_no_beta, max_jobs(dest.size()), dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
            else
                launch_kernel(_cuda_add_cv_to_all_columns, max_jobs(dest.size()), beta, dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_conv(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = A[k]*s[i] + B[k];
            }
        }

        void affine_transform_conv(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src));
            DLIB_CASSERT(have_same_dimensions(A, B));
            DLIB_CASSERT(A.num_samples() == 1 && A.nr() == 1 && A.nc() == 1 && A.k() == src.k());

            launch_kernel(_cuda_affine_transform_conv,max_jobs(dest.size()),
                    dest.device(), src.device(), src.size(), A.device(), B.device(), src.nr()*src.nc(), src.k());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _set_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] = val;
        }

        void set_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_set_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _scale_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] *= val;
        }

        void scale_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_scale_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    }
}

