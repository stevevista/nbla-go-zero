// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/** Add2
 */

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/add2.hpp>
#include <nbla/cuda/function/bc_add2.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void Add2CudaCudnn<T>::setup_impl(const Variables &inputs,
                                  const Variables &outputs) {
  if (inputs[0]->shape() != inputs[1]->shape()) {
    // Trying to fallback to broadcastable Add2.
    Context cuda_ctx = this->ctx_;
    cuda_ctx.set_compute_backend("default");
    this->fall_back_func_ = create_BcAdd2(cuda_ctx);
    this->fall_back_func_->setup(inputs, outputs);
    return;
  }

  Add2<T>::setup_impl(inputs, outputs);
  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, inputs[0]->size()));
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, outputs[0]->size()));
}

template <typename T>
void Add2CudaCudnn<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x0 = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *x1 = inputs[1]->get_data_pointer<T>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  T alpha = 1;
  T beta = 1;
  if (x0 == y) {
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x1,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x1, &beta,
                                    output_desc_, y));
#endif
  } else if (x1 == y) {
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x0,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x0, &beta,
                                    output_desc_, y));
#endif
  } else {
    Add2Cuda<T>::forward_impl(inputs, outputs);
  }
}

template <typename T>
void Add2CudaCudnn<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  T *dx0 = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  T *dx1 = inputs[1]->cast_grad_and_get_pointer<T>(this->ctx_);
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  T alpha = 1;

  if (dx0 != dy && propagate_down[0]) {
    T beta = accum[0] ? 1 : 0;
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, dy,
                                    &beta, output_desc_, dx0));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, dy, &beta,
                                    output_desc_, dx0));
#endif
  }
  if (dx1 != dy && propagate_down[1]) {
    T beta = accum[1] ? 1 : 0;
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, dy,
                                    &beta, output_desc_, dx1));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, dy, &beta,
                                    output_desc_, dx1));
#endif
  }
}

template class Add2CudaCudnn<float>;
}
