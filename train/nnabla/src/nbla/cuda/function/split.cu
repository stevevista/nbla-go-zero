#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/split.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void forward_split_kernel(const int num, const int num_outputs_,
                                     const int outer_size_,
                                     const int inner_size_, const int i0,
                                     const T *x, T *y) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    const int i1 = idx / inner_size_;
    const int i2 = idx % inner_size_;
    y[i1 * inner_size_ + i2] =
        x[i1 * (inner_size_ * num_outputs_) + i0 * inner_size_ + i2];
  }
}

template <typename T>
void SplitCuda<T>::forward_impl(const Variables &inputs,
                                const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  for (int i0 = 0; i0 < this->num_outputs_; ++i0) {
    T *y = outputs[i0]->cast_data_and_get_pointer<T>(this->ctx_);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        forward_split_kernel, this->inner_size_ * this->outer_size_,
        this->num_outputs_, this->outer_size_, this->inner_size_, i0, x, y);
  }
}

template <typename T, bool accum>
__global__ void backward_split_kernel(const int num, const int num_outputs_,
                                      const int outer_size_,
                                      const int inner_size_, const int i0,
                                      T *dx, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    const int i1 = idx / inner_size_;
    const int i2 = idx % inner_size_;
    T &ref = dx[i1 * (inner_size_ * num_outputs_) + i0 * inner_size_ + i2];
    ref = (accum ? ref : 0) + dy[i1 * inner_size_ + i2];
  }
}

template <typename T>
void SplitCuda<T>::backward_impl(const Variables &inputs,
                                 const Variables &outputs,
                                 const vector<bool> &propagate_down,
                                 const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  for (int i0 = 0; i0 < this->num_outputs_; ++i0) {
    const T *dy = outputs[i0]->get_grad_pointer<T>(this->ctx_);
    if (accum[0]) {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((backward_split_kernel<T, true>),
                                     this->inner_size_ * this->outer_size_,
                                     this->num_outputs_, this->outer_size_,
                                     this->inner_size_, i0, dx, dy);
    } else {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((backward_split_kernel<T, false>),
                                     this->inner_size_ * this->outer_size_,
                                     this->num_outputs_, this->outer_size_,
                                     this->inner_size_, i0, dx, dy);
    }
  }
}

// template instantiation
template class SplitCuda<float>;
}
