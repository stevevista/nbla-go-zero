#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// dropout.cu

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/dropout.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_dropout_forward(const int size, const T scale, const T p,
                                       const T *x, T *y, T *m) {
  NBLA_CUDA_KERNEL_LOOP(s, size) {
    m[s] = (m[s] > p) ? 1 : 0;
    y[s] = x[s] * m[s] * scale;
  }
}

template <typename T, bool accum>
__global__ void kernel_dropout_backward(const int size, const T scale,
                                        const T *dy, const T *m, T *dx) {
  NBLA_CUDA_KERNEL_LOOP(s, size) {
    dx[s] = (accum ? dx[s] : 0) + dy[s] * m[s] * scale;
  }
}

template <typename T>
void DropoutCuda<T>::setup_impl(const Variables &inputs,
                                const Variables &outputs) {
  outputs[0]->reshape(inputs[0]->shape(), true);
  this->mask_.reshape(inputs[0]->shape(), true);
}

template <class T>
void DropoutCuda<T>::forward_impl(const Variables &inputs,
                                  const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  Variable &mask = this->mask_;
  T *m = mask.cast_data_and_get_pointer<T>(this->ctx_);
  curand_generate_rand<T>(curand_generator_, 0.0f, 1.0f, m, inputs[0]->size());
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_dropout_forward, inputs[0]->size(),
                                 this->scale_, this->p_, x, y, m);
}

template <class T>
void DropoutCuda<T>::backward_impl(const Variables &inputs,
                                   const Variables &outputs,
                                   const vector<bool> &propagate_down,
                                   const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  Variable &mask = this->mask_;
  const T *m = mask.get_data_pointer<T>(this->ctx_);
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_dropout_backward<T, true>),
                                   inputs[0]->size(), this->scale_, dy, m, dx);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_dropout_backward<T, false>),
                                   inputs[0]->size(), this->scale_, dy, m, dx);
  }
}

template class DropoutCuda<float>;
}
