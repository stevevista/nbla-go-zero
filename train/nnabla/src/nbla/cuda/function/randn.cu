// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/randn.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void RandnCuda<T>::setup_impl(const Variables &inputs,
                              const Variables &outputs) {
  Randn<T>::setup_impl(inputs, outputs);
}

template <typename T>
void RandnCuda<T>::forward_impl(const Variables &inputs,
                                const Variables &outputs) {
  cuda_set_device(device_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  curand_generate_randn<T>(curand_generator_, this->mu_, this->sigma_, y,
                           outputs[0]->size());
}

template <typename T>
void RandnCuda<T>::backward_impl(const Variables &inputs,
                                 const Variables &outputs,
                                 const vector<bool> &propagate_down,
                                 const vector<bool> &accum) {
  // Pass
}

// template instantiation
template class RandnCuda<float>;
}
