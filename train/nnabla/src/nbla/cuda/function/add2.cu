#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// add2.cu

#include <algorithm>
#include <cmath>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/add2.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_add2_forward(const int num, T *y, const T *x0,
                                    const T *x1) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = x0[idx] + x1[idx]; }
}

template <typename T, bool accum>
__global__ void kernel_add2_backward(const int num, T *d, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { d[idx] = (accum ? d[idx] : 0) + dy[idx]; }
}

template <class T>
void Add2Cuda<T>::forward_impl(const Variables &inputs,
                               const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x0 = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *x1 = inputs[1]->get_data_pointer<T>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_add2_forward, size, y, x0, x1);
}

template <class T>
void Add2Cuda<T>::backward_impl(const Variables &inputs,
                                const Variables &outputs,
                                const vector<bool> &propagate_down,
                                const vector<bool> &accum) {
  if (!(propagate_down[0] || propagate_down[1]))
    return;
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      T *dx = inputs[i]->cast_grad_and_get_pointer<T>(this->ctx_);
      if (dx != dy) {
        if (accum[i]) {
          NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_add2_backward<T, true>), size,
                                         dx, dy);
        } else {
          NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_add2_backward<T, false>), size,
                                         dx, dy);
        }
      }
    }
  }
}

// Template instantiation
template class Add2Cuda<float>;
}
