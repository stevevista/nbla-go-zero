// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/** Add2
 */

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/add2.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void Add2CudaCudnn<T>::setup_impl(const Variables &inputs,
                                  Variable* output) {

  Add2<T>::setup_impl(inputs, output);
  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, inputs[0]->size()));
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, output->size()));
}

template <typename T>
void Add2CudaCudnn<T>::forward_impl(const Variables &inputs,
                                    Variable* output) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x0 = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *x1 = inputs[1]->get_data_pointer<T>(this->ctx_);
  T *y = output->cast_data_and_get_pointer<T>(this->ctx_);
  T alpha = 1;
  T beta = 1;

#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x1,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x1, &beta,
                                    output_desc_, y));
#endif
  } else if (x1 == y) {
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x0,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x0, &beta,
                                    output_desc_, y));
#endif

}

template class Add2CudaCudnn<float>;
}
