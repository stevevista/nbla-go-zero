#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/variable.hpp>

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/batch_normalization.hpp>
#include <nbla/cuda/limits.hpp>

namespace nbla {


/******************************************************************************/
/***                 Forward Global Kernel Implementation                     */
/******************************************************************************/

template <typename T>
__global__ void forward_global_kernel(const int size102_, const int size0_,
                                      const int size1_, const int size2_,
                                      const int size02_, const int size12_,
                                      const T *x, 
                                      const T *gamma, const T *beta, T *y) {
  NBLA_CUDA_KERNEL_LOOP(idx, size102_) {
    const int i1 = idx / size02_;
    const int i0 = (idx / size2_) % size0_;
    const int i2 = idx % size2_;
    const int i = i0 * size12_ + i1 * size2_ + i2;
    y[i] = x[i] * gamma[i1] + beta[i1];
  }
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl(const Variables &inputs,
                                             Variable* output) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  forward_impl_global(inputs, output);
  
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl_global(const Variables &inputs,
                                                    Variable* output) {
  // Inputs
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *beta = inputs[1]->get_data_pointer<T>(this->ctx_);
  const T *gamma = inputs[2]->get_data_pointer<T>(this->ctx_);
  // Output
  T *y = output->cast_data_and_get_pointer<T>(this->ctx_);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      forward_global_kernel, this->size1_ * this->size02_, this->size0_,
      this->size1_, this->size2_, this->size02_, this->size12_,
      x, gamma, beta, y);
}


template class BatchNormalizationCuda<float>;
}
