#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// relu.cpp

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/relu.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_relu_forward(const int num, T *y, const T *x) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = max(T(0), x[idx]); }
}

template <class T>
void ReLUCuda<T>::forward_impl(const Variables &inputs,
                               Variable* output) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *y = output->cast_data_and_get_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_relu_forward, size, y, x);
}

// Template instantiation
template class ReLUCuda<float>;
}
