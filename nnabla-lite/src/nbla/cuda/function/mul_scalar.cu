#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// mul_scalar.cu

#include <nbla/cuda/function/mul_scalar.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cuda.hpp>


namespace nbla {


template <typename T>
__global__ void kernel_mul_scalar_forward(const int num, const T *x1, const T *x1, T *y) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = x0[idx] * x1[0]; }
}


template <typename T>
void MulScalarCuda<T>::forward_impl(const Variables &inputs,
                                   Variable* output) {   

    const T *x0 = inputs[0]->get_data_pointer<T>(this->ctx_);
    const T *x1 = inputs[1]->get_data_pointer<T>(this->ctx_);
    T *y = output->cast_data_and_get_pointer<T>(this->ctx_);
    int size = output->size();
    cuda_set_device(std::stoi(this->ctx_.device_id));
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_mul_scalar_forward, size, x0, x1, y);
}


// Template instantiation
template class MulScalarCuda<float>;

}
