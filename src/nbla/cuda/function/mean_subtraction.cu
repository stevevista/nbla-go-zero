#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/mean_subtraction.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void MeanSubtractionCuda<T>::forward_impl(const Variables &inputs,
                                          const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->update_runing_mean_) { // Training mode.
    forward_impl_batch(inputs, outputs);
  } else { // Testing mode.
    forward_impl_global(inputs, outputs);
  }
}

template <typename T>
__global__ void kernel_mean_subtraction_inc_t(T *t, const int max) {
  if (t[0] < max) {
    t[0] = t[0] + 1;
  }
}

template <typename T>
__global__ void kernel_mean_subtraction_forward_batch(const int size1_,
                                                      const int size0_,
                                                      const T *x, T *m, T *rm,
                                                      T *y, const int *t) {
  NBLA_CUDA_KERNEL_LOOP(i1, size1_) {
    T coef = 1.0 / ((*t) + 1);

    // Batch mean
    T mean = 0;
    for (int i0 = 0; i0 < size0_; ++i0) {
      mean += x[i1 + i0 * size1_];
    }
    m[i1] = mean / size0_;

    // Moving mean
    rm[i1] = rm[i1] + (m[i1] - rm[i1]) * coef;

    // Output
    for (int i0 = 0; i0 < size0_; ++i0) {
      y[i1 + i0 * size1_] = x[i1 + i0 * size1_] - rm[i1];
    }
  }
}

template <class T>
void MeanSubtractionCuda<T>::forward_impl_batch(const Variables &inputs,
                                                const Variables &outputs) {
  // Inputs
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  // Output
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  Variable *batch_mean = &this->mean_;
  T *m = batch_mean->cast_data_and_get_pointer<T>(this->ctx_); // batch mean

  // Inputs/Outputs
  T *rm = inputs[1]->cast_data_and_get_pointer<T>(this->ctx_); // running mean
  int *t =
      inputs[2]->cast_data_and_get_pointer<int>(this->ctx_); // running count

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_mean_subtraction_forward_batch,
                                 this->size1_, this->size0_, x, m, rm, y, t);

  kernel_mean_subtraction_inc_t<<<1, 1>>>(t, std::numeric_limits<int>::max());
}

template <typename T>
__global__ void
kernel_mean_subtraction_forward_global(const int size1_, const int size0_,
                                       const T *x, const T *rm, T *y) {
  NBLA_CUDA_KERNEL_LOOP(i1, size1_) {
    for (int i0 = 0; i0 < size0_; ++i0) {
      y[i1 + i0 * size1_] = x[i1 + i0 * size1_] - rm[i1];
    }
  }
}

template <class T>
void MeanSubtractionCuda<T>::forward_impl_global(const Variables &inputs,
                                                 const Variables &outputs) {
  // Inputs
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *rm = inputs[1]->get_data_pointer<T>(this->ctx_); // running mean

  // Output
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_mean_subtraction_forward_global,
                                 this->size1_, this->size0_, x, rm, y);
}

template <typename T>
void MeanSubtractionCuda<T>::backward_impl(const Variables &inputs,
                                           const Variables &outputs,
                                           const vector<bool> &propagate_down,
                                           const vector<bool> &accum) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->update_runing_mean_) { // Training mode.
    backward_impl_batch(inputs, outputs, propagate_down, accum);
  } else { // Testing mode.
    backward_impl_global(inputs, outputs, propagate_down, accum);
  }
}

template <typename T, bool accum>
__global__ void
kernel_mean_subtraction_backward_batch(const int num, T *dx, const T *dy,
                                       const int *t, const int size0_) {
  const T factor = (T)1.0 / ((*t) * size0_);
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    dx[idx] = (accum ? dx[idx] : 0) + dy[idx] * (1 - factor);
  }
}

template <class T>
void MeanSubtractionCuda<T>::backward_impl_batch(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }

  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const int *t = inputs[2]->get_data_pointer<int>(this->ctx_);
  size_t size = inputs[0]->size();
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        (kernel_mean_subtraction_backward_batch<T, true>), size, dx, dy, t,
        this->size0_);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        (kernel_mean_subtraction_backward_batch<T, false>), size, dx, dy, t,
        this->size0_);
  }
}

template <typename T, bool accum>
__global__ void kernel_mean_subtraction_backward_global(const int num, T *dx,
                                                        const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { dx[idx] = (accum ? dx[idx] : 0) + dy[idx]; }
}

template <class T>
void MeanSubtractionCuda<T>::backward_impl_global(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }

  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        (kernel_mean_subtraction_backward_global<T, true>), size, dx, dy);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        (kernel_mean_subtraction_backward_global<T, false>), size, dx, dy);
  }
}

// template instantiation
template class MeanSubtractionCuda<float>;
}
