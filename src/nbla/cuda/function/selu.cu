#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/selu.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_selu_forward(const int num, const T scale_, const T coef,
                                    T *y, const T *x) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    y[idx] = x[idx] > (T)0 ? scale_ * x[idx] : coef * (std::exp(x[idx]) - (T)1);
  }
}

template <typename T, bool accum = true>
__global__ void kernel_selu_backward(const int num, const T scale_,
                                     const T coef, T *dx, const T *x,
                                     const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    dx[idx] =
        (accum ? dx[idx] : (T)0) +
        (x[idx] > (T)0 ? dy[idx] * scale_ : dy[idx] * coef * std::exp(x[idx]));
  }
}

template <typename T>
void SELUCuda<T>::setup_impl(const Variables &inputs,
                             const Variables &outputs) {
  cuda_set_device(this->device_);
  SELU<T>::setup_impl(inputs, outputs);
}

template <typename T>
void SELUCuda<T>::forward_impl(const Variables &inputs,
                               const Variables &outputs) {
  cuda_set_device(this->device_);
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  const T coef = this->alpha_ * this->scale_;
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_selu_forward, size, this->scale_, coef,
                                 y, x);
}

template <typename T>
void SELUCuda<T>::backward_impl(const Variables &inputs,
                                const Variables &outputs,
                                const vector<bool> &propagate_down,
                                const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(this->device_);
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  size_t size = inputs[0]->size();
  const T coef = this->alpha_ * this->scale_;
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_selu_backward<T, true>), size,
                                   this->scale_, coef, dx, x, dy);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_selu_backward<T, false>), size,
                                   this->scale_, coef, dx, x, dy);
  }
}

// template instantiation
template class SELUCuda<float>;
}
