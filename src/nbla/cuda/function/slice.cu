#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////
/* UNDER REVIEW.

   NOTE: hipMemcpy and kernel execution bat setup_impl.
*/
////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/slice.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

__global__ void kernel_slice_create_table(const int num, const int dim,
                                          int *addr_table_buf,
                                          const int *shape_info_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    int addr = 0;
    for (int id = 0; id < dim; id++) {
      const int shape_info_offset = id * 5;
      const int o = (idx / shape_info_buf[shape_info_offset + 1]) // stride_y
                    % shape_info_buf[shape_info_offset];          // shape_y
      const int i = shape_info_buf[shape_info_offset + 3]         // start
                    + o * shape_info_buf[shape_info_offset + 4];  // step
      addr += i * shape_info_buf[shape_info_offset + 2];          // stride_x
    }
    addr_table_buf[idx] = addr;
  }
}

template <typename T>
void SliceCuda<T>::setup_impl(const Variables &inputs,
                              const Variables &outputs) {
  Slice<T>::setup_impl(inputs, outputs);

  // Prepare address table
  const Shape_t shape_y = outputs[0]->shape();
  const Shape_t stride_y = outputs[0]->strides();
  const Shape_t stride_x = inputs[0]->strides();
  size_t size = outputs[0]->size();
  this->addr_table_.reshape(shape_y, true);
  const int shape_info_size = shape_y.size() * 5;
  // out_size, out_stride, in_stride, start, step
  int *shape_info = new int[shape_info_size];
  for (int i = 0; i < shape_y.size(); i++) {
    shape_info[i * 5] = shape_y[i];
    shape_info[i * 5 + 1] = stride_y[i];
    shape_info[i * 5 + 2] = stride_x[i];
    shape_info[i * 5 + 3] = this->start_[0][i];
    shape_info[i * 5 + 4] = this->step_[0][i];
  }
  Shape_t shape_info_shape;
  shape_info_shape.push_back(shape_info_size);
  Variable shape_info_variable;
  shape_info_variable.reshape(shape_info_shape, true);
  int *shape_info_buf =
      shape_info_variable.cast_data_and_get_pointer<int>(this->ctx_);
  hipMemcpy(shape_info_buf, shape_info, sizeof(int) * shape_info_size,
             hipMemcpyHostToDevice);
  delete[] shape_info;
  Variable *addr_table_ = &this->addr_table_;
  int *addr_table_buf = addr_table_->cast_data_and_get_pointer<int>(this->ctx_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_slice_create_table, size,
                                 shape_y.size(), addr_table_buf,
                                 shape_info_buf);
}

template <typename T>
__global__ void kernel_slice_forward(const int num, T *y, const T *x,
                                     const int *addr_table_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = x[addr_table_buf[idx]]; }
}

template <typename T>
void SliceCuda<T>::forward_impl(const Variables &inputs,
                                const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  const int *addr_table_buf =
      this->addr_table_.get_data_pointer<int>(this->ctx_);
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  size_t size = outputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_slice_forward, size, y, x,
                                 addr_table_buf);
}

template <typename T>
__global__ void kernel_slice_backward(const int num, T *dx, const T *dy,
                                      const int *addr_table_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { dx[addr_table_buf[idx]] += dy[idx]; }
}

template <typename T>
void SliceCuda<T>::backward_impl(const Variables &inputs,
                                 const Variables &outputs,
                                 const vector<bool> &propagate_down,
                                 const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (!accum[0])
    inputs[0]->grad()->zero(); // TODO: optimize?
  T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
  const int *addr_table_buf =
      this->addr_table_.get_data_pointer<int>(this->ctx_);
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  size_t size = outputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_slice_backward, size, dx, dy,
                                 addr_table_buf);
}

// template instantiation
template class SliceCuda<float>;
}
