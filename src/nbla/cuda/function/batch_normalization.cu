#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/variable.hpp>

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/batch_normalization.hpp>
#include <nbla/cuda/limits.hpp>

//#include <nbla/cuda/function/kernel/batch_normalization.cuh>
#include "kernel/batch_normalization.cu"

#define BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION

namespace nbla {

template <typename T>
void BatchNormalizationCuda<T>::setup_impl(const Variables &inputs,
                                           const Variables &outputs) {
  BatchNormalization<T>::setup_impl(inputs, outputs);
  v_dmean_.reshape(Shape_t{this->size1_}, true);
  v_dvar_.reshape(Shape_t{this->size1_}, true);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  // setup for transpose
  const int ndim = inputs[0]->ndim();

  // for transpose
  v_axes_.reshape(Shape_t{ndim}, true);
  v_in_strides_.reshape(Shape_t{ndim}, true);
  v_out_strides_.reshape(Shape_t{ndim}, true);
  v_in_shape_.reshape(Shape_t{ndim}, true);
  v_out_shape_.reshape(Shape_t{ndim}, true);
  v_din_trans_.reshape(inputs[0]->shape(), true);

  // work memory for data of each axis
  v_inv_sqrt_variance_.reshape(Shape_t{this->size1_}, true);
  v_t_.reshape(Shape_t{this->size1_}, true);

  // work memory for each block data of shuffle reduction
  this->blocks =
      min((this->size02_ + NBLA_CUDA_NUM_THREADS - 1) / NBLA_CUDA_NUM_THREADS,
          1024);
  v_mean_reduction_space_.reshape(Shape_t{blocks}, true);
  v_variance_reduction_space_.reshape(Shape_t{blocks}, true);
  v_tmp_reduction_space_.reshape(Shape_t{blocks}, true);

  // make shape for transpose
  Context cpu; // CPU Context
  int *p_axes = v_axes_.cast_data_and_get_pointer<int>(cpu);
  int *p_in_strides = v_in_strides_.cast_data_and_get_pointer<int>(cpu);
  int *p_out_strides = v_out_strides_.cast_data_and_get_pointer<int>(cpu);
  int *p_out_shape = v_out_shape_.cast_data_and_get_pointer<int>(cpu);
  int *p_in_shape = v_in_shape_.cast_data_and_get_pointer<int>(cpu);
  for (int i = 0; i < ndim; p_axes[i] = i, ++i)
    ;
  if (this->axes_[0] != 0) {
    p_axes[0] = this->axes_[0];
    p_axes[this->axes_[0]] = 0;
  }
  Shape_t shape(ndim);
  for (int i = 0; i < ndim; ++i)
    shape[i] = inputs[0]->shape()[p_axes[i]];
  v_in_trans_.reshape(shape, true);
  for (int i = 0; i < ndim; ++i) {
    p_in_strides[i] = inputs[0]->strides()[i];
    p_out_strides[i] = v_in_trans_.strides()[i];
    p_in_shape[i] = inputs[0]->shape()[i];
    p_out_shape[i] = v_in_trans_.shape()[i];
  }
#endif
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl(const Variables &inputs,
                                             const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->batch_stat_) { // Training mode.
    forward_impl_batch(inputs, outputs);
  } else { // Testing mode.
    forward_impl_global(inputs, outputs);
  }
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl_batch(const Variables &inputs,
                                                   const Variables &outputs) {
  // Check whether it outputs batch mean and var.
  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;
  if (outputs.size() == 3) {
    batch_mean = outputs[1];
    batch_var = outputs[2];
  }
  // Inputs
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *beta = inputs[1]->get_data_pointer<T>(this->ctx_);
  const T *gamma = inputs[2]->get_data_pointer<T>(this->ctx_);
  // Output
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);
  T *m = batch_mean->cast_data_and_get_pointer<T>(this->ctx_); // batch mean
  T *v = batch_var->cast_data_and_get_pointer<T>(this->ctx_);  // batch varf
  // Inputs/Outputs
  T *rm = inputs[3]->cast_data_and_get_pointer<T>(this->ctx_); // running mean
  T *rv = inputs[4]->cast_data_and_get_pointer<T>(this->ctx_); // running var

#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  const int ndim = inputs[0]->ndim();
  auto get_ = [this](Variable &var) {
    return var.get_data_pointer<int>(this->ctx_);
  };
  auto get_data_ptr_ = [this](Variable &var) {
    return var.cast_data_and_get_pointer<T>(this->ctx_);
  };
  const int *axes = get_(this->v_axes_);
  const int *in_strides = get_(this->v_in_strides_);
  const int *out_strides = get_(this->v_out_strides_);
  const int *in_shape = get_(this->v_in_shape_);
  const int *out_shape = get_(this->v_out_shape_);
  T *in_trans = get_data_ptr_(this->v_in_trans_);
  T *mean_reduction_space = get_data_ptr_(this->v_mean_reduction_space_);
  T *variance_reduction_space =
      get_data_ptr_(this->v_variance_reduction_space_);
  T *inv_sqrt_variance = get_data_ptr_(this->v_inv_sqrt_variance_);
  forward_batch_parallel_reduction(
      this->size0_, this->size1_, this->size2_, ndim, axes, in_strides,
      in_shape, out_strides, out_shape, this->decay_rate_, this->eps_, x, gamma,
      beta, in_trans, m, v, rm, rv, y, mean_reduction_space,
      variance_reduction_space, inv_sqrt_variance);
#else
  forward_batch(this->size0_, this->size1_, this->size2_, this->decay_rate_,
                this->eps_, x, gamma, beta, m, v, rm, rv, y);
#endif
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl_global(const Variables &inputs,
                                                    const Variables &outputs) {
  // Inputs
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  const T *beta = inputs[1]->get_data_pointer<T>(this->ctx_);
  const T *gamma = inputs[2]->get_data_pointer<T>(this->ctx_);
  const T *rm = inputs[3]->get_data_pointer<T>(this->ctx_); // running mean
  const T *rv = inputs[4]->get_data_pointer<T>(this->ctx_); // running var
  // Output
  T *y = outputs[0]->cast_data_and_get_pointer<T>(this->ctx_);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      forward_global_kernel, this->size1_ * this->size02_, this->size0_,
      this->size1_, this->size2_, this->size02_, this->size12_,
      this->decay_rate_, this->eps_, x, rm, rv, gamma, beta, y);
}

template <class T>
void BatchNormalizationCuda<T>::backward_impl(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {

  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->batch_stat_) { // Training mode.
    backward_impl_batch(inputs, outputs, propagate_down, accum);
  } else { // Testing mode.
    NBLA_ERROR(error_code::not_implemented, "");
  }
}

template <class T>
void BatchNormalizationCuda<T>::backward_impl_batch(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!(propagate_down[0] || propagate_down[1] || propagate_down[2])) {
    return;
  }
  // Check whether it outputs batch mean/var.
  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;
  if (outputs.size() == 3) {
    batch_mean = outputs[1];
    batch_var = outputs[2];
  }
  // Commont inputs wrt. gradient.
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  const T *m = batch_mean->get_data_pointer<T>(this->ctx_);
  const T *v = batch_var->get_data_pointer<T>(this->ctx_);
  const T *x = inputs[0]->get_data_pointer<T>(this->ctx_);
  auto get_data_ptr_ = [this](Variable &var) {
    return var.cast_data_and_get_pointer<T>(this->ctx_);
  };
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  int ndim = inputs[0]->ndim();
  auto get_ = [this](Variable &var) {
    return var.get_data_pointer<int>(this->ctx_);
  };
  const int *axes = get_(this->v_axes_);
  const int *in_strides = get_(this->v_in_strides_);
  const int *out_strides = get_(this->v_out_strides_);
  const int *in_shape = get_(this->v_in_shape_);
  const int *out_shape = get_(this->v_out_shape_);
  T *d_x_trans = get_data_ptr_(this->v_in_trans_);
  T *d_dy_trans = get_data_ptr_(this->v_din_trans_);
  T *mean_reduction_space = get_data_ptr_(this->v_mean_reduction_space_);
  T *variance_reduction_space =
      get_data_ptr_(this->v_variance_reduction_space_);
  T *inv_sqrt_variance = get_data_ptr_(this->v_inv_sqrt_variance_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      transpose_2value_kernel, this->size1_ * this->size02_, ndim, axes,
      in_strides, out_strides, out_shape, x, dy, d_x_trans, d_dy_trans);
#endif
  if (propagate_down[0]) {
    if (!accum[0])
      inputs[0]->grad()->zero(); // TODO: optimize this out if possible
    T *dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_);
    const T *g = inputs[2]->get_data_pointer<T>(this->ctx_);
    const T *dm = nullptr;
    const T *dv = nullptr;
    if (outputs.size() == 3) {
      dm = batch_mean->get_grad_pointer<T>(this->ctx_);
      dv = batch_var->get_grad_pointer<T>(this->ctx_);
    }
    T *dmean = get_data_ptr_(this->v_dmean_);
    T *dvar = get_data_ptr_(this->v_dvar_);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
    T *tmp_reduction_space = get_data_ptr_(this->v_tmp_reduction_space_);
    T *t = get_data_ptr_(this->v_t_);
    backward_batch_data_parallel_reduction(
        this->size0_, this->size1_, this->size2_, ndim, axes, in_strides,
        in_shape, out_strides, out_shape, this->decay_rate_, this->eps_, dy, m,
        v, x, g, dm, dv, dx, mean_reduction_space, variance_reduction_space,
        tmp_reduction_space, dmean, dvar, t, inv_sqrt_variance, d_x_trans,
        d_dy_trans);
#else
    backward_batch_data(this->size0_, this->size1_, this->size2_,
                        this->decay_rate_, this->eps_, dy, m, v, x, g, dm, dv,
                        dx, dmean, dvar);
#endif
  }
  if (propagate_down[1] || propagate_down[2]) { // beta and gamma
    NBLA_CHECK(propagate_down[1] && propagate_down[2], error_code::value,
               "'need_grad' of beta and gamma must be the same.");
    if (!accum[1])
      inputs[1]->grad()->zero(); // TODO: optimize this out if possible
    if (!accum[2])
      inputs[2]->grad()->zero(); // TODO: optimize this out if possible
    T *db = inputs[1]->cast_grad_and_get_pointer<T>(this->ctx_);
    T *dg = inputs[2]->cast_grad_and_get_pointer<T>(this->ctx_);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
    backward_batch_gamma_beta_parallel_reduction(
        this->size0_, this->size1_, this->size2_, d_dy_trans, m, v, d_x_trans,
        this->eps_, db, dg, mean_reduction_space, variance_reduction_space,
        inv_sqrt_variance);
#else
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        backward_batch_gamma_beta_kernel, this->size1_, this->size2_,
        this->size02_, this->size12_, this->eps_, dy, m, v, x, db, dg);
#endif
  }
}

template class BatchNormalizationCuda<float>;
}
