#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/function/mean.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/cuda/utils/block_reduce.cuh>

namespace nbla {

template <typename T>
__global__ void kernel_reduce_per_block(const int N, const T *x, T *buff,
                                        T scale = 1) {
  T thread_data = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data += x[i]; }
  thread_data = blockReduceSum(thread_data);
  if (threadIdx.x == 0) {
    buff[blockIdx.x] = thread_data * scale;
  }
}

template <typename T>
void MeanCuda<T>::forward_impl_reduce(const T *x, T *y, int outer_size,
                                      int reduction_size) {
  cuda_set_device(this->device_);
  if (outer_size == 1) {
    if (reduction_size >= 1024) {
      int blocks =
          min(NBLA_CUDA_GET_BLOCKS(reduction_size), /*max blocks*/ 1024);
      shared_ptr<CudaCachedArray> arr_buff =
          make_shared<CudaCachedArray>(blocks, get_dtype<T>(), this->ctx_);
      T *buff = arr_buff->pointer<T>();
      kernel_reduce_per_block<<<blocks, NBLA_CUDA_NUM_THREADS>>>(reduction_size,
                                                                 x, buff);
      kernel_reduce_per_block<<<1, 1024>>>(blocks, buff, y,
                                           (T)(1. / reduction_size));
    } else {
      kernel_reduce_per_block<<<1, 1024>>>(reduction_size, x, y,
                                           (T)(1. / reduction_size));
    }
    return;
  }
  const T *ones = static_cast<const T *>(SingletonManager::get<NNabla>()->ones(
      reduction_size, get_dtype<T>(), this->ctx_));
  cuda_gemv(this->device_, y, x, reduction_size, outer_size, true, ones,
            reduction_size, (T)(1. / reduction_size), (T)0);
}

template <typename T, bool accum>
__global__ void kernel_reduce_mean_backward(const int num, T *dx, const T *dy,
                                            T scale) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    dx[idx] = (accum ? dx[idx] : 0) + scale * (*dy);
  }
}

template <typename T>
void MeanCuda<T>::backward_impl_reduce(const T *dy, T *dx, int outer_size,
                                       int reduction_size, bool accum) {
  cuda_set_device(this->device_);
  if (outer_size == 1) {
    if (accum) {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_mean_backward<T, true>),
                                     reduction_size, dx, dy,
                                     (T)(1. / reduction_size));
    } else {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_mean_backward<T, false>),
                                     reduction_size, dx, dy,
                                     (T)(1. / reduction_size));
    }
    return;
  }
  const T *ones = static_cast<const T *>(SingletonManager::get<NNabla>()->ones(
      reduction_size, get_dtype<T>(), this->ctx_));
  cuda_gemm<T>(this->device_, dx, true, dy, outer_size, 1, false, ones, 1,
               reduction_size, false, (T)(1. / reduction_size),
               (T)(accum ? 1 : 0));
}

// template instantiation
template class MeanCuda<float>;
}
