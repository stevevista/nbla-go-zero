#include "hip/hip_runtime.h"
// Copyright (C) 2015  Davis E. King (davis@dlib.net)
// License: Boost Software License   See LICENSE.txt for the full license.

#include "cuda_utils.h"
#include "cuda_dlib.h"


namespace dlib 
{ 
    namespace cuda 
    {

    // -----------------------------------------------------------------------------------

        void set_device (
            int dev
        )
        {
            CHECK_CUDA(hipSetDevice(dev));
        }

        int get_device (
        )
        {
            int dev = 0;
            CHECK_CUDA(hipGetDevice(&dev));
            return dev;
        }

        std::string get_device_name (
            int device
        )
        {
            hipDeviceProp_t props;
            CHECK_CUDA(hipGetDeviceProperties(&props, device));
            return props.name;
        }

        void set_current_device_blocking_sync(
        )
        {
            CHECK_CUDA(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
        }

        int get_num_devices (
        )
        {
            int num_devices;
            CHECK_CUDA(hipGetDeviceCount(&num_devices));
            return num_devices;
        }

        bool can_access_peer (int device_id, int peer_device_id)
        {
            int can_access;
            CHECK_CUDA(hipDeviceCanAccessPeer(&can_access, device_id, peer_device_id));
            return can_access != 0;
        }
        bool can_access_peer (const tensor& device, const tensor& peer_device)
        {
            return can_access_peer(device.device_id(), peer_device.device_id());
        }

        void device_synchronize (int dev) 
        { 
            raii_set_device set_dev(dev);
            CHECK_CUDA(hipDeviceSynchronize());
        }
        void device_synchronize (const tensor& dev) { device_synchronize(dev.device_id()); }

        enable_peer_access::
        enable_peer_access(
            int device_id,
            int peer_device_id
        ) : call_disable(false), device_id(device_id), peer_device_id(peer_device_id)
        {
            raii_set_device set_dev(device_id);

            auto err = hipDeviceEnablePeerAccess(peer_device_id, 0);
            if (err == hipSuccess)
            {
                call_disable = true;
            }
            else if (err == hipErrorPeerAccessAlreadyEnabled)
            {
                // call hipGetLastError() to dispose of this error since we don't
                // care.
                auto err2 = hipGetLastError();
                if (err2 != hipErrorPeerAccessAlreadyEnabled)
                    CHECK_CUDA(err2);
            }
            else
            {
                CHECK_CUDA(err);
            }
        }


        enable_peer_access::
        ~enable_peer_access() noexcept(false)
        {
            if (call_disable)
            {
                raii_set_device set_dev(device_id);
                CHECK_CUDA(hipDeviceDisablePeerAccess(peer_device_id));
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_columns(float* out, const float* m, const float* v, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = m[j]*v[j%nc];
            }
        }

        void scale_columns (
            tensor& out,
            const tensor& m,
            const tensor& v
        )
        {
            launch_kernel(_cuda_scale_columns, max_jobs(m.size()), out.device(), m.device(), v.device(), m.num_samples(), m.size()/m.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_rows(float* out, const float* m, const float* v, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = m[j]*v[j/nc];
            }
        }

        void scale_rows (
            tensor& out,
            const tensor& m,
            const tensor& v
        )
        {
            launch_kernel(_cuda_scale_rows, max_jobs(m.size()), out.device(), m.device(), v.device(), m.num_samples(), m.size()/m.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_scale_rows2(float* out, const float* m1, const float* m2, const float* v1, const float* v2, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = (m1[j] - m2[j]*v1[j/nc]) * v2[j/nc];
            }
        }

        __global__ void _cuda_scale_rows2_beta(const float beta, float* out, const float* m1, const float* m2, const float* v1, const float* v2, size_t nr, size_t nc)
        {
            for (auto j : grid_stride_range(0, nr*nc))
            {
                out[j] = beta*out[j] + (m1[j] - m2[j]*v1[j/nc]) * v2[j/nc];
            }
        }

        void scale_rows2 (
            float beta, 
            tensor& out,
            const tensor& m1,
            const tensor& m2,
            const tensor& v1,
            const tensor& v2
        )
        {
            if (beta == 0)
            {
                launch_kernel(_cuda_scale_rows2, max_jobs(m1.size()), out.device(),
                    m1.device(), m2.device(), v1.device(), v2.device(), m1.num_samples(),
                    m1.size()/m1.num_samples());
            }
            else
            {
                launch_kernel(_cuda_scale_rows2_beta, max_jobs(m1.size()), beta,
                    out.device(), m1.device(), m2.device(), v1.device(), v2.device(),
                    m1.num_samples(), m1.size()/m1.num_samples());
            }
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_exp(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::exp(src[i]);
        }

        void exp (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_exp, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_log(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::log(src[i]);
        }

        void log (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_log, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_log10(float* dest, const float* src, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
                dest[i] = ::log10(src[i]);
        }

        void log10 (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_ASSERT(dest.size() == src.size());
            launch_kernel(_cuda_log10, max_jobs(src.size()), dest.device(), src.device(), src.size());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_multiply1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = 0;
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i%s1_n]*s2[i%s2_n];
            }
        }

        __global__ void _cuda_multiply1_add_to(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i]*s2[i];
            }
        }
        __global__ void _cuda_multiply2_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n, size_t max_size)
        {
            for (auto i : grid_stride_range(0, n))
            {
                for (size_t j = i; j < max_size; j += n)
                    d[i] += s1[j%s1_n]*s2[j%s2_n];
            }
        }

        __global__ void _cuda_multiply3_add_to(float* d, const float* s1, const float* s2, 
                                       size_t n, size_t s1_n, size_t s2_n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i%s1_n]*s2[i%s2_n];
            }
        }

        void multiply (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {

            DLIB_CASSERT(dest.k() == src1.k() && src1.k() == src2.k() &&
                dest.nr() == src1.nr() && src1.nr() == src2.nr() &&
                dest.nc() == src1.nc() && src1.nc() == src2.nc() );
            const long MD = std::max(std::max(dest.num_samples(),src1.num_samples()),src2.num_samples());
            DLIB_CASSERT((dest.num_samples()==1 || dest.num_samples()==MD) &&
                (src1.num_samples()==1 || src1.num_samples()==MD) &&
                (src2.num_samples()==1 || src2.num_samples()==MD) );

            if (dest.size() == 0)
                return;

            const size_t max_size = std::max(std::max(dest.size(),src1.size()),src2.size());
            const auto d = dest.host();
            const auto s1 = src1.host();
            const auto s2 = src2.host();
            if (dest.size() == src1.size() && src1.size() == src2.size())
            {
                if (add_to)
                    launch_kernel(_cuda_multiply1_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
                else
                    launch_kernel(_cuda_multiply1,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), src1.size());
            }
            else if (dest.num_samples() == 1)
            {
                if (add_to)
                    launch_kernel(_cuda_multiply2_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
                else
                    launch_kernel(_cuda_multiply2,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size(), max_size);
            }
            else
            {
                if (add_to)
                    launch_kernel(_cuda_multiply3_add_to,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
                else
                    launch_kernel(_cuda_multiply3,max_jobs(dest.size()),dest.device(), src1.device(), src2.device(), 
                                                dest.size(), src1.size(), src2.size());
            }
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_mult1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]*s2[i];
            }
        }

        __global__ void _cuda_mult1_add_to(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += s1[i]*s2[i];
            }
        }

        __global__ void _cuda_mult2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1*v2;
            }
        }

        __global__ void _cuda_mult2_add_to(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] += v1*v2;
            }
        }

        void multiply_zero_padded (
            bool add_to,
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                if (add_to)
                    launch_kernel(_cuda_mult1_add_to,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
                else
                    launch_kernel(_cuda_mult1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                if (add_to)
                {
                    // Otherwise, do the more complex version with bounds checking.
                    launch_kernel(_cuda_mult2_add_to,max_jobs(dest.size()),
                                dest.device(), src1.device(), src2.device(), 
                                dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                                src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                                src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                                );
                }
                else
                {
                    // Otherwise, do the more complex version with bounds checking.
                    launch_kernel(_cuda_mult2,max_jobs(dest.size()),
                                dest.device(), src1.device(), src2.device(), 
                                dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                                src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                                src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                                );
                }
            }
        }

    // ------------------------------------------------------------------------------------

        __global__ void _cuda_add1(float* d, const float* s1, const float* s2, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = s1[i]+s2[i];
            }
        }

        __global__ void _cuda_add2(float* d, const float* s1, const float* s2, 
                                   size_t dn, size_t dk, size_t dr, size_t dc,
                                   size_t s1n, size_t s1k, size_t s1r, size_t s1c,
                                   size_t s2n, size_t s2k, size_t s2r, size_t s2c)
        {
            for (auto i : grid_stride_range(0, dn*dk*dr*dc))
            {
                size_t n,k,r,c;
                unpack_idx(i, dk,dr,dc, n,k,r,c);

                float v1 = 0;
                float v2 = 0;

                if (n < s1n &&
                    k < s1k &&
                    r < s1r &&
                    c < s1c )
                {
                    v1 = s1[pack_idx(s1k,s1r,s1c, n,k,r,c)];
                }

                if (n < s2n &&
                    k < s2k &&
                    r < s2r &&
                    c < s2c )
                {
                    v2 = s2[pack_idx(s2k,s2r,s2c, n,k,r,c)];
                }

                d[i] = v1+v2;
            }
        }

        void add (
            tensor& dest,
            const tensor& src1,
            const tensor& src2
        )
        {
            if (dest.size() == 0)
                return;

            // Do the simple and fast version if everything has the same dimensions
            if (have_same_dimensions(dest, src1) &&
                have_same_dimensions(dest, src2))
            {
                launch_kernel(_cuda_add1,max_jobs(dest.size()), dest.device(), src1.device(), src2.device(), dest.size());
            }
            else
            {
                // Otherwise, do the more complex version with bounds checking.
                launch_kernel(_cuda_add2,max_jobs(dest.size()),
                            dest.device(), src1.device(), src2.device(), 
                            dest.num_samples(), dest.k(), dest.nr(), dest.nc(),
                            src1.num_samples(), src1.k(), src1.nr(), src1.nc(),
                            src2.num_samples(), src2.k(), src2.nr(), src2.nc()
                            );
            }

        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_scaled(float* d, const float* s, size_t n, float scale)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] += scale*s[i]; 
            }
        }

        void add_scaled(
            tensor& dest,
            const float scale,
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.size()==src.size());
            launch_kernel(_cuda_add_scaled,max_jobs(dest.size()),dest.device(), src.device(), dest.size(), scale);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_add_cv_to_all_columns(float beta, float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = beta*dest[i] + alpha*src[i/stride];
            }
        }

        __global__ void _cuda_add_cv_to_all_columns_no_beta(float* dest, float alpha, const float* src, size_t size, size_t stride)
        {
            for (auto i : grid_stride_range(0, size))
            {
                dest[i] = alpha*src[i/stride];
            }
        }

        void add_cv_to_all_columns(
            float beta, 
            tensor& dest, 
            float alpha, 
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.num_samples() == src.num_samples() && src.num_samples() == src.size());
            if (beta == 0)
                launch_kernel(_cuda_add_cv_to_all_columns_no_beta, max_jobs(dest.size()), dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
            else
                launch_kernel(_cuda_add_cv_to_all_columns, max_jobs(dest.size()), beta, dest.device(), alpha, src.device(), dest.size(), dest.size()/dest.num_samples());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_range(
            float* d, const float* s1, const float* s2, const float* s3, size_t begin, size_t end, float A, float B, float C
        )
        {
            for (auto i : grid_stride_range(begin, end))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i];
            }
        }


        void affine_transform_range(
            size_t begin,
            size_t end,
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size());
            DLIB_CASSERT(dest.size()==src2.size());
            DLIB_CASSERT(dest.size()==src3.size());
            DLIB_CASSERT(begin <= end && end <= dest.size());
            launch_kernel(_cuda_affine_transform_range,max_jobs(end-begin),
                dest.device(), src1.device(),
                src2.device(), src3.device(), begin, end, A, B, C);
        }


    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_compute_adam_update(
            size_t begin,
            size_t end,
            float* s,
            float* m,
            float* v,
            const float alpha,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const float* params,
            const float* params_grad
        )
        {
            const float eps = 1e-8;
            // The loop is equivalent to doing this:
            //   m = momentum1*m + (1-momentum1)    *   (weight_decay*params + params_grad);
            //   v = momentum2*v + (1-momentum2)*squared(weight_decay*params + params_grad);
            //   s = -alpha*m/(sqrt(v) + eps);
            for (auto i : grid_stride_range(begin, end))
            {
                float g = (weight_decay*params[i] + params_grad[i]);
                m[i] = momentum1*m[i] + (1-momentum1)*g;
                v[i] = momentum2*v[i] + (1-momentum2)*g*g;
                s[i] = -alpha*m[i]/(std::sqrt(v[i]) + eps);
            }
        }

        void compute_adam_update (
            size_t begin,
            size_t end,
            tensor& s,
            tensor& m,
            tensor& v,
            const float t,
            const float learning_rate,
            const float weight_decay,
            const float momentum1,
            const float momentum2,
            const tensor& params,
            const tensor& params_grad
        )
        {
            DLIB_CASSERT(s.size() == m.size() &&
                         s.size() == v.size() &&
                         s.size() == params.size() &&
                         s.size() == params_grad.size());
            DLIB_CASSERT(begin <= end && end <= params.size());
            const float alpha = learning_rate*std::sqrt(1-std::pow(momentum2,t))/(1-std::pow(momentum1, t));

            launch_kernel(_cuda_compute_adam_update,max_jobs(end-begin),
                    begin, end, s.device(), m.device(), v.device(), alpha, weight_decay,
                    momentum1, momentum2, params.device(), params_grad.device());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform_conv(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs, size_t ks)
        {
            for (auto i : grid_stride_range(0, n))
            {
                auto k = (i/bs)%ks;
                d[i] = A[k]*s[i] + B[k];
            }
        }

        void affine_transform_conv(
            tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(have_same_dimensions(dest, src));
            DLIB_CASSERT(have_same_dimensions(A, B));
            DLIB_CASSERT(A.num_samples() == 1 && A.nr() == 1 && A.nc() == 1 && A.k() == src.k());

            launch_kernel(_cuda_affine_transform_conv,max_jobs(dest.size()),
                    dest.device(), src.device(), src.size(), A.device(), B.device(), src.nr()*src.nc(), src.k());
        }


    // ----------------------------------------------------------------------------------------

        __global__ void _set_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] = val;
        }

        void set_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_set_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _scale_tensor(float* out, size_t n, const float val)
        {
            for (auto i : grid_stride_range(0, n))
                out[i] *= val;
        }

        void scale_tensor (
            tensor& t,
            float value
        )
        {
            launch_kernel(_scale_tensor, max_jobs(t.size()), t.device(), t.size(), value);
        }

    // -----------------------------------------------------------------------------------
    // -----------------------------------------------------------------------------------

        __global__ void _cuda_threshold(float* d, size_t n, float thresh)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = d[i]>thresh ? 1:0;
            }
        }

        void threshold (
            tensor& data,
            float thresh
        )
        {
            launch_kernel(_cuda_threshold,max_jobs(data.size()),data.device(), data.size(), thresh);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_prelu(const float* s, float* d, size_t n, const float* pp)
        {
            const float p = *pp;
            for (auto i : grid_stride_range(0, n))
            {
                if (s[i] > 0)
                    d[i] = s[i];
                else
                    d[i] = p*s[i];
            }
        }

        void prelu (
            tensor& dest,
            const tensor& src,
            const tensor& param
        )
        {
            launch_kernel(_cuda_prelu, max_jobs(dest.size()), 
                src.device(), dest.device(), src.size(), param.device());
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_copy_tensor_add_to (float* dest, size_t size,  const float* src,  size_t dest_stride, size_t src_stride, size_t block_size)
        {
            for(auto i : grid_stride_range(0, size)) 
            {
                size_t blk = i/block_size;
                size_t j = i%block_size;
                dest[blk*dest_stride + j] += src[blk*src_stride + j];
            }
        }

        __global__ void _cuda_copy_tensor (float* dest, size_t size,  const float* src,  size_t dest_stride, size_t src_stride, size_t block_size)
        {
            for(auto i : grid_stride_range(0, size)) 
            {
                size_t blk = i/block_size;
                size_t j = i%block_size;
                dest[blk*dest_stride + j] = src[blk*src_stride + j];
            }
        }

        void copy_tensor(
            bool add_to,
            tensor& dest,
            size_t dest_k_offset,
            const tensor& src,
            size_t src_k_offset,
            size_t count_k
        )
        {
            const size_t dest_sample_size = static_cast<size_t>(dest.nc() * dest.nr() * dest.k());
            const size_t src_sample_size = static_cast<size_t>(src.nc() * src.nr() * src.k());

            const size_t block_size = count_k * dest.nc() * dest.nr();

            DLIB_CASSERT(dest.num_samples() == src.num_samples() &&
                         dest.nc() == src.nc() && dest.nr() == src.nr(), "All sources should fit into dest tensor size");
            DLIB_CASSERT(dest.k() - dest_k_offset >= count_k, "Not enough space in dest tensor");
            DLIB_CASSERT(src.k() - src_k_offset >= count_k, "Not enough space in src tensor");

            float* dest_p = dest.device() + dest_k_offset * dest.nc() * dest.nr();
            const float* src_p = src.device() + src_k_offset * src.nc() * src.nr();;

            if (add_to)
            {
                launch_kernel(_cuda_copy_tensor_add_to, max_jobs(dest.size()), 
                              dest_p, block_size*dest.num_samples(),
                              src_p, dest_sample_size, src_sample_size, block_size);
            }
            else
            {
                launch_kernel(_cuda_copy_tensor, max_jobs(dest.size()), 
                              dest_p, block_size*dest.num_samples(),
                              src_p, dest_sample_size, src_sample_size, block_size);
            }
        }

    // ----------------------------------------------------------------------------------------

    }
}

